#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"
#include <cstdio>

#ifdef __HIPCC__
__global__ void dummy_kernel(int *out) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *out = 42;
    }
}
#endif

int launch_dummy_kernel() {
#ifdef __HIPCC__
    int *d_out = nullptr;
    int h_out = 0;
    hipError_t err = hipMalloc(&d_out, sizeof(int));
    if (err != hipSuccess) {
        std::fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    dummy_kernel<<<1, 32>>>(d_out);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        hipFree(d_out);
        return 2;
    }
    err = hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(err));
        hipFree(d_out);
        return 3;
    }
    hipFree(d_out);
    // Silent success for now; could print h_out if needed
    return h_out == 42 ? 0 : 4;
#else
    return 0; // CUDA disabled; treat as success stub.
#endif
}
